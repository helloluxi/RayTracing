#include "hip/hip_runtime.h"

#include "raytracing.cu"
#include <iostream>
#include <jsoncpp/json/json.h>
#include <string>
#include <vector>

#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image_write.h"

using namespace std;

const float gammaCorrector = 2.2f;
const float invGamma = 1.0f / gammaCorrector;

void LoadTexture(Texture*& tex, const char* path);
void LoadObjModel(TriangleMesh* mesh, string& path);
void readFloat3(Json::Value& json, string key, float3& v);
void readFloat(Json::Value& json, string key, float& f);
void readInt(Json::Value& json, string key, int& i);
void readString(Json::Value& json, string key, string& s);
#define parseInt(name) int name; readInt(root, #name, name);
#define parseFloat(name) float name; readFloat(root, #name, name);
#define parseFloat3(name) float3 name; readFloat3(root, #name, name);
#define parseString(name) string name; readString(root, #name, name);
#define invalidParameter(name) catch(...){ cerr << "Invalid parameter \"" << name << "\" at line " << __LINE__ << endl; exit(1); }

int main(int argc, char** argv)
{
    auto awakeTime = clock();

    // read config
    Json::Value root;
    {
        string inputPath = argc == 1 ? "config.json" : argv[1];
        ifstream f;
        f.open(inputPath);
        if(!f.is_open()) { 
            cerr << "Cannot find file: " << inputPath << endl; 
            exit(1); 
        }
        Json::Reader reader;
        if(!reader.parse(f, root))
        {
            cerr << "Parse Error: " << inputPath << endl;
            exit(1);
        }
    }

    parseInt(myThreadNum)
    parseInt(screenWidth)
    parseInt(screenHeight)
    parseInt(nSample)
    parseInt(maxDepth)
    parseFloat(verticalFOV)
    parseFloat3(cameraPos)
    parseFloat3(cameraFront)
    parseFloat3(cameraUp)
    parseFloat3(cameraRight)
    parseString(outputPath)

    int pixelNum = screenWidth * screenHeight;
    int myGridNum = (pixelNum + myThreadNum - 1) / myThreadNum;
    

    // init random seed
    hiprandState* rd;
    checkCudaErrors(hipMalloc((void**)&rd, pixelNum * sizeof(hiprandState)));
    runInit(rd, myGridNum, myThreadNum);
    checkCudaErrors(hipDeviceSynchronize());


    // init scene
    Scene scene;
    try{
        auto node = root["materials"];
        if(node.isNull()){
            scene.materialCount = 0;
        }
        else{
            scene.materialCount = node.size();
            checkCudaErrors(hipMallocManaged(&scene.materials, sizeof(Shader) * scene.materialCount));
            for (int i = 0; i < scene.materialCount; i++)
            {
                readFloat3(node[i], "albedo", scene.materials[i].albedo);
                readFloat(node[i], "metallic", scene.materials[i].metallic);
                readFloat(node[i], "roughness", scene.materials[i].roughness);
            }
        }
    } invalidParameter("materials")
    try{
        auto node = root["planes"];
        if(node.isNull()){
            scene.planeCount = 0;
        }
        else{
            scene.planeCount = node.size();
            checkCudaErrors(hipMallocManaged(&scene.planes, sizeof(Plane) * scene.planeCount));
            for (int i = 0; i < scene.planeCount; i++)
            {
                readFloat3(node[i], "point", scene.planes[i].point);
                readFloat3(node[i], "normal", scene.planes[i].normal);
                readInt(node[i], "materialId", scene.planes[i].materialId);
            }
        }
    } invalidParameter("planes")
    try{
        auto node = root["spheres"];
        if(node.isNull()){
            scene.sphereCount = 0;
        }
        else{
            scene.sphereCount = node.size();
            checkCudaErrors(hipMallocManaged(&scene.spheres, sizeof(Sphere) * scene.sphereCount));
            for (int i = 0; i < scene.sphereCount; i++)
            {
                readFloat3(node[i], "center", scene.spheres[i].center);
                readFloat(node[i], "r", scene.spheres[i].r);
                readInt(node[i], "materialId", scene.spheres[i].materialId);
            }
        }
    } invalidParameter("spheres")
    try{
        auto node = root["meshes"];
        if(node.isNull()){
            scene.meshCount = 0;
        }
        else{
            scene.meshCount = node.size();
            checkCudaErrors(hipMallocManaged(&scene.meshes, sizeof(TriangleMesh) * scene.meshCount));
            for (int i = 0; i < scene.meshCount; i++)
            {
                string objPath;
                readString(node[i], "path", objPath);
                LoadObjModel(&scene.meshes[i], objPath);
                readInt(node[i], "materialId", scene.meshes[i].materialId);
            }
        }
    } invalidParameter("meshes")
    try{
        auto node = root["areaLight"];
        if(!node.isNull()){
            scene.useLightSampling = true;
            checkCudaErrors(hipMallocManaged(&scene.light, sizeof(AreaLight)));
            readFloat3(node, "center", scene.light->center);
            readFloat3(node, "intensity", scene.light->intensity);
            readFloat3(node, "normal", scene.light->normal);
            readFloat3(node, "right", scene.light->right);
            readFloat3(node, "up", scene.light->up);
            readFloat(node, "size", scene.light->size);
        }
    } invalidParameter("areaLight")


    // Malloc result image
    float3* result;
    checkCudaErrors(hipMallocManaged(&result, sizeof(float3) * pixelNum));


    // Ray Tracing
    auto startTime = clock();
    printf("Initializing Time = %.3fs\n", (float)(startTime - awakeTime) / CLOCKS_PER_SEC);
    runKernel(result, rd, scene,
        cameraPos, cameraFront, cameraUp, cameraRight,
        screenWidth, screenHeight, nSample, maxDepth, verticalFOV / screenHeight,
        myGridNum, myThreadNum);
    checkCudaErrors(hipDeviceSynchronize());
    printf("Ray Tracing Time = %.3fs\n", (float)(clock() - startTime) / CLOCKS_PER_SEC);


    // Write image to file
    stbi_uc output[pixelNum * 3];
    for (size_t i = 0; i < pixelNum; i++)
    {
        output[i * 3] = (stbi_uc)(powf(clamp(result[i].x, 0.0f, 1.0f), invGamma) * 255.f);
        output[i * 3 + 1] = (stbi_uc)(powf(clamp(result[i].y, 0.0f, 1.0f), invGamma) * 255.f);
        output[i * 3 + 2] = (stbi_uc)(powf(clamp(result[i].z, 0.0f, 1.0f), invGamma) * 255.f);
    }
    stbi_write_png(outputPath.c_str(), screenWidth, screenHeight, 3, output, screenWidth * 3);


    // Free
    checkCudaErrors(hipDeviceReset());
    return 0;
}

void LoadTexture(Texture*& tex, const char* path) {
    checkCudaErrors(hipMallocManaged(&tex, sizeof(Texture)));
    auto img = stbi_load(path, &tex->size[0], &tex->size[1], &tex->size[2], 3);
    if (img == NULL) {
        printf("Image %s not found!", path);
        exit(1);
    }
    checkCudaErrors(hipMallocManaged(&tex->texture, sizeof(float3) * tex->size[0] * tex->size[1]));
    for (size_t i = 0; i < tex->size[0] * tex->size[1]; i++)
    {
        tex->texture[i] = make_float3(powf(img[i * 3] / 255.f, gammaCorrector),
            powf(img[i * 3 + 1] / 255.f, gammaCorrector), powf(img[i * 3 + 2] / 255.f, gammaCorrector));
    }
    stbi_image_free(img);
}
void readInt(Json::Value& json, string key, int& i){
    try{
        i = json[key].asInt();
    }catch(...){
        cerr << "Invalid parameter \"" << key << "\"." << endl;
        exit(1);
    }
}
void readFloat(Json::Value& json, string key, float& f){
    try{
        f = json[key].asFloat();
    }catch(...){
        cerr << "Invalid parameter \"" << key << "\"." << endl;
        exit(1);
    }
}
void readFloat3(Json::Value& json, string key, float3& v){
    try{
        v.x = json[key][0].asFloat();
        v.y = json[key][1].asFloat();
        v.z = json[key][2].asFloat();
    }catch(...){
        cerr << "Invalid parameter \"" << key << "\"." << endl;
        exit(1);
    }
}
void readString(Json::Value& json, string key, string& s){
    try{
        s = json[key].asString();
    }catch(...){
        cerr << "Invalid parameter \"" << key << "\"." << endl;
        exit(1);
    }
}
void LoadObjModel(TriangleMesh* mesh, string& path){
    vector<float3> vertices, normals;
    vector<int> triangles, triangleNormals;

    ifstream f;
    f.open(path);
    if(!f.is_open()) { 
        cerr << "Cannot find file: " << path << endl; 
        exit(1); 
    }

    try{
        char c;
        while(!f.eof()){
            f >> c;
            if(c == 'v'){
                if(f.get() == 'n'){
                    float3 v;
                    f >> c >> v.x >> v.y >> v.z;
                    normals.emplace_back(v);
                }
                else{
                    float3 v;
                    f >> v.x >> v.y >> v.z;
                    vertices.emplace_back(v);
                }
            }
            else if(c == 'f'){
                int t, n;
                f >> t >> c >> c >> n;
                triangles.push_back(t - 1);
                triangleNormals.push_back(n - 1);
                f >> t >> c >> c >> n;
                triangles.push_back(t - 1);
                triangleNormals.push_back(n - 1);
                f >> t >> c >> c >> n;
                triangles.push_back(t - 1);
                triangleNormals.push_back(n - 1);
            }
            else{
                string s;
                getline(f, s);
                // cout << "Ignore line: " << s << endl;
            }
        }
    }
    catch(...){
        cerr << "Cannot parse file: " << path << endl;
        exit(1);
    }
        
    mesh->vertexCount = vertices.size();
    mesh->triangleCount = triangles.size() / 3;
    checkCudaErrors(hipMalloc(&mesh->vertices, sizeof(float3) * vertices.size()));
    checkCudaErrors(hipMemcpy(mesh->vertices, &vertices[0], vertices.size() * sizeof(float3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&mesh->normals, sizeof(float3) * normals.size()));
    checkCudaErrors(hipMemcpy(mesh->normals, &normals[0], normals.size() * sizeof(float3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&mesh->triangles, sizeof(int) * triangles.size()));
    checkCudaErrors(hipMemcpy(mesh->triangles, &triangles[0], triangles.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&mesh->triangleNormals, sizeof(int) * triangleNormals.size()));
    checkCudaErrors(hipMemcpy(mesh->triangleNormals, &triangleNormals[0], triangleNormals.size() * sizeof(int), hipMemcpyHostToDevice));
}

